#include "hip/hip_runtime.h"
//This is code write to run on the CPU but written just like you would write it for the GPU 
//and write to compiled with NVCC, It also uses the full force matrix.																																											
// nvcc nbodyCPU2.cu -o CPU2 -lglut -lm -lGLU -lGL
//To stop hit "control c" in the window you launched it from.
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 100

#define XWindowSize 2500
#define YWindowSize 2500

#define DRAW 10
#define PRINT 100
#define DAMP .5

#define G 1.0
#define H 1.0

#define DT 0.0001

#define EYE 10.0
#define FAR 50.0

// Globals
float4 p[N];
float3 v[N], f[N];
float STOP_TIME; 
FILE *data_file, *data_file1, *data_file2;

void set_initail_conditions()
{
	int i,j,k,num,particles_per_side;
    float position_start, temp;
    float initail_seperation;
	
	STOP_TIME = 100.0;
   
	temp = pow((float)N,1.0/3.0) + 0.99999;
	particles_per_side = temp;
	printf("\n cube root of N = %d \n", particles_per_side);
    position_start = -(particles_per_side -1.0)/2.0;
	initail_seperation = 2.0;
	
	for(i=0; i<N; i++)
	{
		p[i].w = 1.0;
	}
	
	num = 0;
	for(i=0; i<particles_per_side; i++)
	{
		for(j=0; j<particles_per_side; j++)
		{
			for(k=0; k<particles_per_side; k++)
			{
			    if(N <= num) break;
				p[num].x = position_start + i*initail_seperation;
				p[num].y = position_start + j*initail_seperation;
				p[num].z = position_start + k*initail_seperation;
				v[num].x = 0.0;
				v[num].y = 0.0;
				v[num].z = 0.0;
				num++;
			}
		}
	}
}

void draw_picture()
{
	int i;
	
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	glColor3d(1.0,1.0,0.5);
	for(i=0; i<N; i++)
	{
		glPushMatrix();
		glTranslatef(p[i].x, p[i].y, p[i].z);
		glutSolidSphere(0.1,20,20);
		glPopMatrix();
	}
	
	glutSwapBuffers();
}
                                 
__host__ __device__ float3 getBodyBodyForce(float4 p0, float4 p1)
{
    float3 f;
    float dx = p1.x - p0.x;
    float dy = p1.y - p0.y;
    float dz = p1.z - p0.z;
    float r2 = dx*dx + dy*dy + dz*dz;
    float r = sqrt(r2);
    
    float force  = (G*p0.w*p1.w)/(r2) - (H*p0.w*p1.w)/(r2*r2);
    
    f.x = force*dx/r;
    f.y = force*dy/r;
    f.z = force*dz/r;
    
    return(f);
}

void n_body()
{
	float3 force_mag; 
	float dt;
	int   tdraw = 0; 
	int   tprint = 0;
	float time = 0.0;
	int i,j;
	
	dt = DT;

	while(time < STOP_TIME)
	{
		for(i=0; i<N; i++)
		{
			f[i].x = 0.0;
			f[i].y = 0.0;
			f[i].z = 0.0;
		}
		
		for(i=0; i<N; i++)
		{
			for(j=0; j<N; j++)
			{	
				if(i != j) 
				{
				    force_mag = getBodyBodyForce(p[i], p[j]);
				    f[i].x += force_mag.x;
				    f[i].y += force_mag.y;
				    f[i].z += force_mag.z;
				}
			}
		}

		for(i=0; i<N; i++)
		{
			if(time == 0.0)
			{
				v[i].x += (f[i].x/p[i].w)*0.5*dt;
				v[i].y += (f[i].y/p[i].w)*0.5*dt;
				v[i].z += (f[i].z/p[i].w)*0.5*dt;
			}
			else
			{
				v[i].x += ((f[i].x-DAMP*v[i].x)/p[i].w)*dt;
				v[i].y += ((f[i].y-DAMP*v[i].y)/p[i].w)*dt;
				v[i].z += ((f[i].z-DAMP*v[i].z)/p[i].w)*dt;
			}

			p[i].x += v[i].x*dt;
			p[i].y += v[i].y*dt;
			p[i].z += v[i].z*dt;
		}

		if(tdraw == DRAW) 
		{
			draw_picture();
			tdraw = 0;
		}
		
		time += dt;
		tdraw++;
		tprint++;
	}
}

void control()
{	
	set_initail_conditions();
	draw_picture();
    n_body();
	
	printf("\n DONE \n");
	while(1);
}

void Display(void)
{
	gluLookAt(EYE, EYE, EYE, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	control();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);

	glMatrixMode(GL_PROJECTION);

	glLoadIdentity();

	glFrustum(-0.2, 0.2, -0.2, 0.2, 0.2, FAR);

	glMatrixMode(GL_MODELVIEW);
}

int main(int argc, char** argv)
{
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("2 Body 3D");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMainLoop();
	return 0;
}






